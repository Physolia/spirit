
#include <hip/hip_runtime.h>
#ifdef SPIRIT_USE_CUDA

#include "data/Geometry.hpp"
#include "engine/Vectormath_Defines.hpp"
#include <Spirit_Defines.h>
#include <engine/Hamiltonian_Heisenberg.hpp>
#include <engine/Method_MC.hpp>
#include <engine/Vectormath.hpp>
#include <iostream>
#include <fstream>
#include <utility/Constants.hpp>
#include <curand_kernel.h>
#include <curand.h>
#include <math.h>
#include <Eigen/Dense>
#include <Eigen/Core>
using namespace Utility;

class curandStateWrapper
{
public:
    curandState * state;
};

namespace Engine
{
using namespace Vectormath;

// A helper struct that contains device pointers to the relevant fields
struct Geometry_Device_Ptrs
{
    const int * n_cells;
    const scalar * mu_s;
    const int n_cell_atoms;
    const int * atom_types;

    Geometry_Device_Ptrs( const Data::Geometry & geom )
            : n_cells( geom.n_cells.data() ),
              mu_s( geom.mu_s.data() ),
              n_cell_atoms( geom.n_cell_atoms ),
              atom_types( geom.atom_types.data() )
    {
    }
};

// A helper struct that contains device pointers to the relevant fields
struct Hamiltonian_Device_Ptrs
{
    Geometry_Device_Ptrs geometry;
    const int anisotropy_n_axes;
    const scalar * anisotropy_magnitudes;
    const Vector3 * anisotropy_normals;
    const int * anisotropy_indices;

    const int exchange_n_pairs;
    const Pair * exchange_pairs;
    const scalar * exchange_magnitudes;

    const int dmi_n_pairs;
    const Pair * dmi_pairs;
    const scalar * dmi_magnitudes;
    const Vector3 * dmi_normals;

    const scalar external_field_magnitude;
    const Vector3 external_field_normal;

    const int * boundary_conditions;

    const int idx_zeeman;
    const int idx_anisotropy;
    const int idx_exchange;
    const int idx_dmi;

    Hamiltonian_Device_Ptrs( const Hamiltonian_Heisenberg & ham )
            : geometry( *ham.geometry ),
              anisotropy_n_axes( ham.anisotropy_indices.size() ),
              anisotropy_magnitudes( ham.anisotropy_magnitudes.data() ),
              anisotropy_normals( ham.anisotropy_normals.data() ),
              anisotropy_indices( ham.anisotropy_indices.data() ),
              exchange_n_pairs( ham.exchange_pairs.size() ),
              exchange_pairs( ham.exchange_pairs.data() ),
              exchange_magnitudes( ham.exchange_magnitudes.data() ),
              dmi_n_pairs( ham.dmi_pairs.size() ),
              dmi_pairs( ham.dmi_pairs.data() ),
              dmi_magnitudes( ham.dmi_magnitudes.data() ),
              dmi_normals( ham.dmi_normals.data() ),
              external_field_magnitude( ham.external_field_magnitude ),
              external_field_normal( ham.external_field_normal ),
              boundary_conditions( ham.boundary_conditions.data() ),
              idx_zeeman( ham.Idx_Zeeman() ),
              idx_anisotropy( ham.Idx_Anisotropy() ),
              idx_exchange( ham.Idx_Exchange() ),
              idx_dmi( ham.Idx_DMI() )
    {
    }
};



__device__ scalar Energy_Single_Spin( int ispin, const Vector3 * spins, Hamiltonian_Device_Ptrs ham )
{
    // This function is a replacement for the Hamiltonian_Heisenberg member function of the same name, it can be used from
    // within cuda kernels
    auto & anisotropy_indices  = ham.anisotropy_indices;
    auto & anisotropy_normals  = ham.anisotropy_normals;
    auto & exchange_pairs      = ham.exchange_pairs;
    auto & exchange_magnitudes = ham.exchange_magnitudes;
    auto & dmi_pairs           = ham.dmi_pairs;
    auto & dmi_normals         = ham.dmi_normals;
    auto & dmi_magnitudes      = ham.dmi_magnitudes;
    auto & boundary_conditions = ham.boundary_conditions;
    auto & geometry            = ham.geometry;

    scalar Energy = 0;
    if( cu_check_atom_type( ham.geometry.atom_types[ispin] ) )
    {
        int icell  = ispin / ham.geometry.n_cell_atoms;
        int ibasis = ispin - icell * ham.geometry.n_cell_atoms;
        auto mu_s  = ham.geometry.mu_s;
        Pair pair_inv;

        // External field
        if( ham.idx_zeeman >= 0 )
            Energy -= mu_s[ispin] * ham.external_field_magnitude * ham.external_field_normal.dot( spins[ispin] );

        // Anisotropy
        if( ham.idx_anisotropy >= 0 )
        {
            for( int iani = 0; iani < ham.anisotropy_n_axes; ++iani )
            {
                if( anisotropy_indices[iani] == ibasis )
                {
                    if( cu_check_atom_type( ham.geometry.atom_types[ispin] ) )
                        Energy -= ham.anisotropy_magnitudes[iani]
                                  * powf( anisotropy_normals[iani].dot( spins[ispin] ), 2.0 );
                }
            }
        }

        // Exchange
        if( ham.idx_exchange >= 0 )
        {
            for( unsigned int ipair = 0; ipair < ham.exchange_n_pairs; ++ipair )
            {
                const auto & pair = exchange_pairs[ipair];
                if( pair.i == ibasis )
                {
                    int jspin = cu_idx_from_pair(
                        ispin, boundary_conditions, geometry.n_cells, geometry.n_cell_atoms, geometry.atom_types,
                        pair );
                    if( jspin >= 0 )
                        Energy -= ham.exchange_magnitudes[ipair] * spins[ispin].dot( spins[jspin] );
                }
            }
        }

        // DMI
        if( ham.idx_dmi >= 0 )
        {
            for( unsigned int ipair = 0; ipair < ham.dmi_n_pairs; ++ipair )
            {
                const auto & pair = dmi_pairs[ipair];
                if( pair.i == ibasis )
                {
                    int jspin = cu_idx_from_pair(
                        ispin, boundary_conditions, geometry.n_cells, geometry.n_cell_atoms, geometry.atom_types,
                        pair );
                    if( jspin >= 0 )
                        Energy -= ham.dmi_magnitudes[ipair]
                                  * ham.dmi_normals[ipair].dot( spins[ispin].cross( spins[jspin] ) );
                    
                }
            }
        }

        // TODO: Quadruplets are missing, but can be added later ...
    }
    return Energy;
}


__device__ bool cu_metropolis_spin_trial(
    int ispin, const Vector3 * spins_old, Vector3 * spins_new, Hamiltonian_Device_Ptrs ham, const scalar rng1,
    const scalar rng2, const scalar rng3, const scalar cos_cone_angle, const scalar temperature ) // aditional parameter kB_T needs to be input
{

    // TODO: Implement
    // This function should perform a metropolis spin trial, using the same logic as in core/src/engine/Method_MC.cpp

    // A few things I would like to point out:
    //     1. Remember that pointers that live on the host side have no meaning when used on the device. This also
    //     applies to the `this` pointer of any object that was constructed on the host side. Therefore you will have to
    //     explicitly copy some parameters of the method to the device. An easy way to do this is in the argument list
    //     of the kernel invocation.
    //     2. Any type that is derived from `field` (defined in `core/include/engine/Vectormath_Defines.hpp`), uses a
    //     special allocator so that the pointers you get with the `data()` member method can be used either on the host
    //     or the device. E.g spins.data() is a Vector3 * that can be dereferenced on the host and the device.
    //     3. Only functions that are marked as _device_ can be used within kernels. That means you have to replace
    //     some functions when porting code from the cpu to the gpu.

    // IMP: usman: In meeting, need to ask about the data type ----
    

    Matrix3 local_basis; // usman: Ask in the meeting..., classes with functions in CUDA
    const Vector3 e_z{0,0,1};
    const scalar kB_T = Constants::k_B * temperature; //usman: kB_T Need to be explicitly inputed to the function
    // local_basis = Matrix3::Identity();
    // Calculate local basis for the spin
    
    if(fabs( spins_old[ispin].z() ) < 1 - 1e-10 ) //usman: fabs function from CUDA Math
    {
        local_basis.col( 2 ) = spins_old[ispin];
        local_basis.col( 0 ) = ( local_basis.col( 2 ).cross( e_z ).eval() ).normalized();
        local_basis.col( 1 ) = local_basis.col( 2 ).cross( local_basis.col( 0 ) ).eval();
    } else {
        local_basis = Matrix3::Identity();
    }

    scalar costheta = 1 - ( 1 - cos_cone_angle ) * rng1;
    scalar sintheta = sqrt( 1 - costheta * costheta );

    // Random distribution of phi between 0 and 360 degrees
    scalar phi = 2 * Constants::Pi * rng2;

    Vector3 local_spin_new{ sintheta * cos( phi ), sintheta * sin( phi ), costheta };

    // New spin orientation in regular basis
    spins_new[ispin] = local_basis * local_spin_new;

    // Energy difference of configurations with and without displacement

    scalar Eold = Energy_Single_Spin(ispin, spins_old, ham);
    scalar Enew = Energy_Single_Spin(ispin, spins_new, ham);

    scalar Ediff = Enew - Eold;

    // return;

    // printf("Eold: %f, Enew: %f, Ediff: %f \n", Eold, Enew, Ediff);
    // Metropolis criterion: reject the step if energy rose
    if( Ediff > 1e-14 )
    {
        if( temperature < 1e-12 ) // usman: Need to explicitly define the value of temperature rather than just passing the pointer
        {
            // Restore the spin
            spins_new[ispin] = spins_old[ispin];
            return false;
        }
        else
        {
            // Exponential factor
            scalar exp_ediff = exp(-Ediff / kB_T ); // CUDA Exponential Function

            // Only reject if random number is larger than exponential
            if( exp_ediff < rng3 )
            {
                // Restore the spin
                spins_new[ispin] = spins_old[ispin];
                // Counter for the number of rejections
                return false;
            }
        }
    }
    return true;
}

__global__ void cu_setup_curand(curandState * states)
{
    int seed     = 1234;
    int blockId  = blockIdx.x + blockIdx.y * gridDim.x + gridDim.x * gridDim.y * blockIdx.z;
    int threadId = blockId * (blockDim.x * blockDim.y * blockDim.z) + (threadIdx.z * (blockDim.x * blockDim.y)) + (threadIdx.y * blockDim.x) + threadIdx.x;
    curand_init(seed, threadId, 0, &states[threadId]);
}

void Method_MC::Setup_Curand()
{
    dim3 block(1, 1, 1);
    dim3 grid(max(n_blocks[0]/2,1), max(n_blocks[1]/2,1), max(n_blocks[2]/2,1));

    Log.Send(Utility::Log_Level::Info, Utility::Log_Sender::MC, fmt::format("Performing block decomposition for parallel Metropolis algorithm"));

    dev_random = new curandStateWrapper();

    cudaMalloc((void **) &(dev_random->state), grid.x * grid.y * grid.z * sizeof(curandState));
    cu_setup_curand<<<grid, block>>>( dev_random->state );
}

__global__ void cu_parallel_metropolis(const Vector3 * spins_old, Vector3 * spins_new, Hamiltonian_Device_Ptrs ham, const int * phase, const int * n_blocks, const int * block_size_min, const int * rest, curandState *states, scalar cos_cone_angle, scalar temperature)
{
    // Number of spins
    int nos  = ham.geometry.n_cells[0] * ham.geometry.n_cells[1] * ham.geometry.n_cells[2] * ham.geometry.n_cell_atoms;

    int block_a    = 2 * blockIdx.x + phase[0];
    int block_b    = 2 * blockIdx.y + phase[1];
    int block_c    = 2 * blockIdx.z + phase[2];

    int seed       = 1234;

    int blockId  = blockIdx.x+ blockIdx.y * gridDim.x + gridDim.x * gridDim.y * blockIdx.z;
    int threadId = blockId * (blockDim.x * blockDim.y * blockDim.z) + (threadIdx.z * (blockDim.x * blockDim.y)) + (threadIdx.y * blockDim.x)+ threadIdx.x;

    int block_size_c = (block_c == n_blocks[2] - 1) ? block_size_min[2] + rest[2] : block_size_min[2]; // Account for the remainder of division (n_cells[i] / block_size_min[i]) by increasing the block size at the edges
    int block_size_b = (block_b == n_blocks[1] - 1) ? block_size_min[1] + rest[1] : block_size_min[1];
    int block_size_a = (block_a == n_blocks[0] - 1) ? block_size_min[0] + rest[0] : block_size_min[0];

    // Iterate over the current block (this has to be done serially again)
    for(int cc = 0; cc < block_size_c; cc++)
    {
        for(int bb = 0; bb < block_size_b; bb++)
        {
            for(int aa = 0; aa < block_size_a; aa++)
            {
                for(int ibasis = 0; ibasis < ham.geometry.n_cell_atoms; ibasis++)
                {
                    int a = block_a * block_size_min[0] + aa; // We do not have to worry about the remainder of the division here, it is contained in the 'aa'/'bb'/'cc' offset
                    int b = block_b * block_size_min[1] + bb;
                    int c = block_c * block_size_min[2] + cc;

                    scalar rng1 = curand_uniform(&states[threadId]);
                    scalar rng2 = curand_uniform(&states[threadId]);
                    scalar rng3 = curand_uniform(&states[threadId]);

                    // printf("%f %f %f\n", rng1, rng2, rng3);

                    // Compute the current spin idx
                    int ispin = ibasis + ham.geometry.n_cell_atoms * (a + ham.geometry.n_cells[0] * (b + ham.geometry.n_cells[1] * c));
                    bool test = cu_metropolis_spin_trial(ispin, spins_old, spins_new, ham, rng1, rng2, rng3, cos_cone_angle, temperature);
                }
            }
        }
    }
}

__global__ void cu_metropolis_order( const Vector3 * spins_old, Vector3 * spins_new, int * order, unsigned int * counter, Hamiltonian_Device_Ptrs ham, const int * phase, const int * n_blocks, const int * block_size_min, int * rest, curandState *states)
{
    // TODO: Implement such that the spins are addressed in the right order according to the block decomposition

    int nos  = ham.geometry.n_cells[0] * ham.geometry.n_cells[1] * ham.geometry.n_cells[2] * ham.geometry.n_cell_atoms;

    int block_a    = 2 * blockIdx.x + phase[0];
    int block_b    = 2 * blockIdx.y + phase[1];
    int block_c    = 2 * blockIdx.z + phase[2];

    int seed     = block_a;
    curand_init(seed, block_a, 0, &states[block_a]);

    int i = 0;

    int block_size_c = (block_c == n_blocks[2] - 1) ? block_size_min[2] + rest[2] : block_size_min[2]; // Account for the remainder of division (n_cells[i] / block_size_min[i]) by increasing the block size at the edges
    int block_size_b = (block_b == n_blocks[1] - 1) ? block_size_min[1] + rest[1] : block_size_min[1];
    int block_size_a = (block_a == n_blocks[0] - 1) ? block_size_min[0] + rest[0] : block_size_min[0];

    // Iterate over the current block (this has to be done serially again)
    for(int cc = 0; cc < block_size_c; cc++)
    {
        for(int bb = 0; bb < block_size_b; bb++)
        {
            for(int aa = 0; aa < block_size_a; aa++)
            {
                for(int ibasis = 0; ibasis < ham.geometry.n_cell_atoms; ibasis++)
                {
                    int a = block_a * block_size_min[0] + aa; // We do not have to worry about the remainder of the division here, it is contained in the 'aa'/'bb'/'cc' offset
                    int b = block_b * block_size_min[1] + bb;
                    int c = block_c * block_size_min[2] + cc;

                    // Compute the current spin idx
                    int ispin = ibasis + ham.geometry.n_cell_atoms * (a + ham.geometry.n_cells[0] * (b + ham.geometry.n_cells[1] * c));
                    
                    scalar rng1 = curand_uniform(&states[block_a]);
                    scalar rng2 = curand_uniform(&states[block_a]);
                    scalar rng3 = curand_uniform(&states[block_a]);
                    bool tmp;
                    // cu_metropolis_spin_trial(ispin, spins_old, spins_new, ham, rng1, rng2, rng3, 0.5, 100, 100);
                    
                    // cu_metropolis_spin_trial(
                    // int ispin, Vector3 * spins_old, Vector3 * spins_new, Hamiltonian_Device_Ptrs ham, const scalar rng1,
                    // const scalar rng2, const scalar rng3, const scalar cos_cone_angle, scalar temperature, const scalar kB_T )
                    unsigned int current_count = atomicInc(counter, nos);
                    order[current_count] = ispin;
                    // printf("a %i, b %i, c %i, ispin %d\nblock_a %i, block_b %i, block_c %i \n aa %i, bb %i, cc %i\n--\n", a, b, c, ispin, block_a, block_b, block_c, aa, bb, cc);
                    //printf("a_blocksize %i, b_blocksize %i, c_blocksize %i \n", block_size_a, block_size_b, block_size_c);
                    //
                    //printf("ham.geometry.n_cells[0] %i \n", ham.geometry.n_cells[0]);
                    //printf("ham.geometry.n_cells[1] %i \n", ham.geometry.n_cells[1]);
                    //printf("n_block[0]=%i, n_block[1]=%i, n_block[2]=%i \n", n_blocks[0], n_blocks[1], n_blocks[2]);
                    //printf("block_a=%i, block_b=%i, block_c=%i \n" , block_a, block_b, block_c);
                    //printf("%d \n", i);

                }
            }
        }
    }
}

void Method_MC::Parallel_Metropolis( const vectorfield & spins_old, vectorfield & spins_new )
{
    auto hamiltonian = dynamic_cast<Engine::Hamiltonian_Heisenberg *>( this->systems[0]->hamiltonian.get() );
    auto ham_ptrs    = Hamiltonian_Device_Ptrs(*hamiltonian); // Collect the device pointers in a struct

    // We allocate these two fields to record the order of spin-trials
    auto order       = field<int>(spins_old.size(), -1);
    auto counter     = field<unsigned int>(1, 0);

    dim3 block(1, 1, 1);
    dim3 grid(max(n_blocks[0]/2,1), max(n_blocks[1]/2,1), max(n_blocks[2]/2,1));

    scalar temperature    = this->parameters_mc->temperature;
    scalar cos_cone_angle = std::cos(this->cone_angle);

    int phase_c_lim = n_blocks[2] > 1 ? 2 : 1;
    int phase_b_lim = n_blocks[1] > 1 ? 2 : 1;
    int phase_a_lim = n_blocks[0] > 1 ? 2 : 1;

    for(int phase_c = 0; phase_c < phase_c_lim; phase_c++)
    {
        for(int phase_b = 0; phase_b < phase_b_lim; phase_b++)
        {
            for(int phase_a = 0; phase_a < phase_a_lim; phase_a++)
            {
                const field<int> phase = {phase_a, phase_b, phase_c};
                cu_parallel_metropolis<<<grid, block>>>( spins_old.data(), spins_new.data(), ham_ptrs, phase.data(), n_blocks.data(), block_size_min.data(), rest.data(), dev_random->state, cos_cone_angle, temperature);
            }
        }
    }

    // cu_metropolis_order<<<grid, block>>>( spins_old.data(), spins_new.data(), order.data(), counter.data(), ham_ptrs, 1, 0, 0, rest[0],rest[1], rest[2], 2, 2, 1);  // cu_metropolis_order<<<grid, block>>>( spins_old.data(), spins_new.data(), order.data(), counter.data(), ham_ptrs, phase_a, phase_b, phase_c, rest[0],rest[1], rest[2], block_size_min[0], block_size_min[1], block_size_min[2]);
    cudaDeviceSynchronize();
    return;

    // dump the results in some file
    std::ofstream myfile;
    myfile.open ("mc_access_order.txt");

    myfile << "# a b c idx_spin idx_trial\n";
    // Write out the order
    auto n_cells = this->systems[0]->geometry->n_cells;
    auto n_cell_atoms = this->systems[0]->geometry->n_cell_atoms;
    field<int> tupel = {0,0,0,0};
    for(int i=0; i<order.size(); i++)
    {
        if(order[i] >= 0)
        {
            Vectormath::tupel_from_idx(order[i], tupel, {n_cell_atoms, n_cells[0], n_cells[1], n_cells[2]});
            myfile << tupel[1] << " " << tupel[2] << " " << tupel[3] << " " << order[i] << " " << i << "\n";
        }
    }
    myfile.close();
}

} // namespace Engine

#endif

/*
__device__ void cu_metropolis_spin_trial(
    int ispin, Vector3 * spins_old, Vector3 * spins_new, Hamiltonian_Device_Ptrs ham, const scalar rng1,
    const scalar rng2, const scalar rng3, const scalar cos_cone_angle, scalar temperature )
{

    // TODO: Implement
    // This function should perform a metropolis spin trial, using the same logic as in core/src/engine/Method_MC.cpp

    // A few things I would like to point out:
    //     1. Remember that pointers that live on the host side have *no* meaning when used on the device. This also
    //     applies to the `this` pointer of any object that was constructed on the host side. Therefore you will have to
    //     explicitly copy some parameters of the method to the device. An easy way to do this is in the argument list
    //     of the kernel invocation.
    //     2. Any type that is derived from `field` (defined in `core/include/engine/Vectormath_Defines.hpp`), uses a
    //     special allocator so that the pointers you get with the `data()` member method can be used either on the host
    //     or the device. E.g spins.data() is a Vector3 * that can be dereferenced on the host and the device.
    //     3. Only functions that are marked as __device__ can be used within kernels. That means you have to replace
    //     some functions when porting code from the cpu to the gpu.

    Matrix3 local_basis;
    const Vector3 e_z{0,0,1};
    const scalar kB_T = Constants::k_B * temperature;

    // Calculate local basis for the spin
    if( std::abs(spins_old[ispin].z()) < 1-1e-10 )
    {
        local_basis.col(2) = spins_old[ispin];
        local_basis.col(0) = (local_basis.col(2).cross(e_z)).normalized();
        local_basis.col(1) = local_basis.col(2).cross(local_basis.col(0));
    } else
    {
        local_basis = Matrix3::Identity();
    }

    // Rotation angle between 0 and cone_angle degrees
    scalar costheta = 1 - (1 - cos_cone_angle) * rng1;

    scalar sintheta = std::sqrt(1 - costheta*costheta);

    // Random distribution of phi between 0 and 360 degrees
    scalar phi = 2*Constants::Pi * rng2;

    Vector3 local_spin_new{ sintheta * std::cos(phi),
                            sintheta * std::sin(phi),
                            costheta };

    // New spin orientation in regular basis
    spins_new[ispin] = local_basis * local_spin_new;

    // Energy difference of configurations with and without displacement
    scalar Eold  = this->systems[0]->hamiltonian->Energy_Single_Spin(ispin, spins_old);
    scalar Enew  = this->systems[0]->hamiltonian->Energy_Single_Spin(ispin, spins_new);
    scalar Ediff = Enew-Eold;

    // Metropolis criterion: reject the step if energy rose
    if( Ediff > 1e-14 )
    {
        if( this->parameters_mc->temperature < 1e-12 )
        {
            // Restore the spin
            spins_new[ispin] = spins_old[ispin];
            return false;
        }
        else
        {
            // Exponential factor
            scalar exp_ediff    = std::exp( -Ediff/kB_T );
            // Only reject if random number is larger than exponential
            if( exp_ediff < rng3 )
            {
                // Restore the spin
                spins_new[ispin] = spins_old[ispin];
                // Counter for the number of rejections
                return false;
            }
        }
    }
    return true;
}
*/